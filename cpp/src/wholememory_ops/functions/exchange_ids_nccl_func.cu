#include "exchange_ids_nccl_func.h"

#include <cub/device/device_radix_sort.cuh>
#include <thrust/sequence.h>

#include "error.hpp"
#include "logger.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory_ops/register.hpp"

namespace wholememory_ops {

template<typename IndexT>
void exchange_ids_temp_func(const void* indices_before_sort,
                            wholememory_array_description_t indices_desc,
                            const int64_t* host_recv_rank_id_count_ptr,
                            const int64_t* host_rank_id_count_ptr,
                            const int64_t* host_rank_id_offset_ptr,
                            temp_memory_handle* dev_recv_indice_buffer,
                            void* indices_after_sort,
                            int64_t* raw_indices,
                            wholememory_comm_t wm_comm,
                            wm_thrust_allocator* p_thrust_allocator,
                            hipStream_t stream) {
  auto index_type = indices_desc.dtype;
  WHOLEMEMORY_CHECK(indices_desc.storage_offset == 0);
  WHOLEMEMORY_CHECK(index_type == WHOLEMEMORY_DT_INT || index_type == WHOLEMEMORY_DT_INT64);
  wm_thrust_allocator& allocator = *p_thrust_allocator;

  int64_t *seq_indices = reinterpret_cast<int64_t *>(allocator.allocate(
      wholememory_get_memory_element_count_from_array(&indices_desc) * sizeof(int64_t)));
  thrust::sequence(thrust::cuda::par(allocator).on(stream), seq_indices,
                   seq_indices + indices_desc.size, 0);
  const IndexT* indices_to_sort = static_cast<const IndexT*>(indices_before_sort);
  IndexT* sorted_indice = static_cast<IndexT*>(indices_after_sort);
  void* cub_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(cub_temp_storage,
                                  temp_storage_bytes,
                                  indices_to_sort,
                                  sorted_indice,
                                  seq_indices,
                                  raw_indices,
                                  indices_desc.size,
                                  0,
                                  sizeof(IndexT) * 8,
                                  stream);
  cub_temp_storage = allocator.allocate(temp_storage_bytes);
  hipcub::DeviceRadixSort::SortPairs(cub_temp_storage,
                                  temp_storage_bytes,
                                  indices_to_sort,
                                  sorted_indice,
                                  seq_indices,
                                  raw_indices,
                                  indices_desc.size,
                                  0,
                                  sizeof(IndexT) * 8,
                                  stream);
  int64_t total_recv_count = 0;
  int world_size;
  WHOLEMEMORY_CHECK(wholememory_communicator_get_size(&world_size, wm_comm) == WHOLEMEMORY_SUCCESS);
  std::vector<size_t> host_recv_offset(world_size);
  for (int i = 0; i < world_size; i++) {
    host_recv_offset[i] = total_recv_count;
    total_recv_count += host_recv_rank_id_count_ptr[i];
  }
  IndexT *dev_recv_indice_buffer_ptr =
      static_cast<IndexT *>(dev_recv_indice_buffer->device_malloc(total_recv_count, index_type));
  wm_comm->alltoallv(sorted_indice,
                     dev_recv_indice_buffer_ptr,
                     reinterpret_cast<const size_t*>(host_rank_id_count_ptr),
                     reinterpret_cast<const size_t*>(host_rank_id_offset_ptr),
                     reinterpret_cast<const size_t*>(host_recv_rank_id_count_ptr),
                     host_recv_offset.data(),
                     index_type,
                     stream);
  wm_comm->sync_stream(stream);
  allocator.deallocate(reinterpret_cast<char*>(seq_indices), wholememory_get_memory_size_from_array(&indices_desc));
  allocator.deallocate(static_cast<char*>(cub_temp_storage), temp_storage_bytes);
}

REGISTER_DISPATCH_ONE_TYPE(ExchangeIDsNCCL, exchange_ids_temp_func, SINT3264)

wholememory_error_code_t exchange_ids_func(const void *indices_before_sort,
                                           wholememory_array_description_t indices_desc,
                                           const int64_t *host_recv_rank_id_count_ptr,
                                           const int64_t *host_rank_id_count_ptr,
                                           const int64_t *host_rank_id_offset_ptr,
                                           temp_memory_handle *dev_recv_indices_buffer_handle,
                                           void* indices_after_sort,
                                           int64_t *raw_indices,
                                           wholememory_comm_t wm_comm,
                                           wm_thrust_allocator *p_thrust_allocator,
                                           hipStream_t stream) {
  try {
    DISPATCH_ONE_TYPE(indices_desc.dtype,
                      ExchangeIDsNCCL,
                      indices_before_sort,
                      indices_desc,
                      host_recv_rank_id_count_ptr,
                      host_rank_id_count_ptr,
                      host_rank_id_offset_ptr,
                      dev_recv_indices_buffer_handle,
                      indices_after_sort,
                      raw_indices,
                      wm_comm,
                      p_thrust_allocator,
                      stream);
  } catch (wholememory::cuda_error &wce) {
    WHOLEMEMORY_ERROR("exchange_ids_func CUDA LOGIC Error %s\n", wce.what());
    return WHOLEMEMORY_CUDA_ERROR;
  } catch (wholememory::logic_error &wle) {
    WHOLEMEMORY_ERROR("exchange_ids_func LOGIC Error %s\n", wle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_UNKNOW_ERROR;
  }
  return WHOLEMEMORY_SUCCESS;
}

}  // namespace wholememory_ops