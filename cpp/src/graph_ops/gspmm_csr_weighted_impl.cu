#include "gspmm_csr_weighted_func.cuh"
#include <wholememory/wholememory.h>

#include "wholememory_ops/register.hpp"

namespace graph_ops {
REGISTER_DISPATCH_ONE_TYPE(gSpmmCSRWeightedForward, gspmm_csr_weighted_forward_func, HALF_FLOAT)
REGISTER_DISPATCH_ONE_TYPE(gSpmmCSRWeightedBackward, gspmm_csr_weighted_backward_func, HALF_FLOAT)

wholememory_error_code_t gspmm_csr_weighted_forward_impl(
  int* csr_row_ptr,
  wholememory_array_description_t csr_row_ptr_array_desc,
  int* csr_col_ptr,
  wholememory_array_description_t csr_col_ptr_array_desc,
  void* edge_weight_ptr,
  wholememory_tensor_description_t edge_weight_tensor_desc,
  void* feature_ptr,
  wholememory_tensor_description_t feature_tensor_desc,
  void* output_feature_ptr,
  wholememory_tensor_description_t output_feature_tensor_desc,
  hipStream_t stream)
{
  try {
    DISPATCH_ONE_TYPE(feature_tensor_desc.dtype,
                      gSpmmCSRWeightedForward,
                      csr_row_ptr,
                      csr_row_ptr_array_desc,
                      csr_col_ptr,
                      csr_col_ptr_array_desc,
                      edge_weight_ptr,
                      edge_weight_tensor_desc,
                      feature_ptr,
                      feature_tensor_desc,
                      output_feature_ptr,
                      output_feature_tensor_desc,
                      stream);

  } catch (const wholememory::cuda_error& rle) {
    // WHOLEMEMORY_FAIL_NOTHROW("%s", rle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (const wholememory::logic_error& le) {
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_LOGIC_ERROR;
  }
  return WHOLEMEMORY_SUCCESS;
}

wholememory_error_code_t gspmm_csr_weighted_backward_impl(
  int* csr_row_ptr,
  wholememory_array_description_t csr_row_ptr_array_desc,
  int* csr_col_ptr,
  wholememory_array_description_t csr_col_ptr_array_desc,
  void* edge_weight_ptr,
  wholememory_tensor_description_t edge_weight_tensor_desc,
  void* feature_ptr,
  wholememory_tensor_description_t feature_tensor_desc,
  void* grad_feature_ptr,
  wholememory_tensor_description_t grad_feature_tensor,
  void* output_grad_edge_weight_ptr,
  wholememory_tensor_description_t output_grad_edge_weight_tensor_desc,
  void* output_grad_feature_ptr,
  wholememory_tensor_description_t output_grad_feature_tensor_desc,
  hipStream_t stream)
{
  try {
    DISPATCH_ONE_TYPE(feature_tensor_desc.dtype,
                      gSpmmCSRWeightedBackward,
                      csr_row_ptr,
                      csr_row_ptr_array_desc,
                      csr_col_ptr,
                      csr_col_ptr_array_desc,
                      edge_weight_ptr,
                      edge_weight_tensor_desc,
                      feature_ptr,
                      feature_tensor_desc,
                      grad_feature_ptr,
                      grad_feature_tensor,
                      output_grad_edge_weight_ptr,
                      output_grad_edge_weight_tensor_desc,
                      output_grad_feature_ptr,
                      output_grad_feature_tensor_desc,
                      stream);

  } catch (const wholememory::cuda_error& rle) {
    // WHOLEMEMORY_FAIL_NOTHROW("%s", rle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (const wholememory::logic_error& le) {
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_LOGIC_ERROR;
  }
  return WHOLEMEMORY_SUCCESS;
}

}  // namespace graph_ops
