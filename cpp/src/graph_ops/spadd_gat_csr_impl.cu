/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "spadd_gat_csr_func.cuh"
#include <wholememory/wholememory.h>

#include "wholememory_ops/register.hpp"

namespace graph_ops {

REGISTER_DISPATCH_ONE_TYPE(SpADDGATCSRForward, spadd_gat_csr_forward_func, HALF_FLOAT)
REGISTER_DISPATCH_ONE_TYPE(SpADDGATCSRBackward, spadd_gat_csr_backward_func, HALF_FLOAT)
wholememory_error_code_t spadd_gat_csr_forward_impl(
  int* csr_row_ptr,
  wholememory_array_description_t csr_row_ptr_array_desc,
  int* csr_col_ptr,
  wholememory_array_description_t csr_col_ptr_array_desc,
  void* edge_weight_left_ptr,
  wholememory_matrix_description_t edge_weight_left_matrix_desc,
  void* edge_weight_right_ptr,
  wholememory_matrix_description_t edge_weight_right_matrix_desc,
  void* output_score_ptr,
  wholememory_matrix_description_t output_score_matrix_desc,
  hipStream_t stream)
{
  try {
    DISPATCH_ONE_TYPE(edge_weight_left_matrix_desc.dtype,
                      SpADDGATCSRForward,
                      csr_row_ptr,
                      csr_row_ptr_array_desc,
                      csr_col_ptr,
                      csr_col_ptr_array_desc,
                      edge_weight_left_ptr,
                      edge_weight_left_matrix_desc,
                      edge_weight_right_ptr,
                      edge_weight_right_matrix_desc,
                      output_score_ptr,
                      output_score_matrix_desc,
                      stream);

  } catch (const wholememory::cuda_error& rle) {
    // WHOLEMEMORY_FAIL_NOTHROW("%s", rle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (const wholememory::logic_error& le) {
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_LOGIC_ERROR;
  }
  return WHOLEMEMORY_SUCCESS;
}

wholememory_error_code_t spadd_gat_csr_backward_impl(
  int* csr_row_ptr,
  wholememory_array_description_t csr_row_ptr_array_desc,
  int* csr_col_ptr,
  wholememory_array_description_t csr_col_ptr_array_desc,
  void* grad_score_ptr,
  wholememory_matrix_description_t grad_score_matrix_desc,
  void* output_grad_edge_weight_left_ptr,
  wholememory_matrix_description_t edge_weight_left_matrix_desc,
  void* output_grad_edge_weight_right_ptr,
  wholememory_matrix_description_t edge_weight_right_matrix_desc,
  hipStream_t stream)
{
  try {
    DISPATCH_ONE_TYPE(edge_weight_left_matrix_desc.dtype,
                      SpADDGATCSRBackward,
                      csr_row_ptr,
                      csr_row_ptr_array_desc,
                      csr_col_ptr,
                      csr_col_ptr_array_desc,
                      grad_score_ptr,
                      grad_score_matrix_desc,
                      output_grad_edge_weight_left_ptr,
                      edge_weight_left_matrix_desc,
                      output_grad_edge_weight_right_ptr,
                      edge_weight_right_matrix_desc,
                      stream);

  } catch (const wholememory::cuda_error& rle) {
    // WHOLEMEMORY_FAIL_NOTHROW("%s", rle.what());
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (const wholememory::logic_error& le) {
    return WHOLEMEMORY_LOGIC_ERROR;
  } catch (...) {
    return WHOLEMEMORY_LOGIC_ERROR;
  }
  return WHOLEMEMORY_SUCCESS;
}

}  // namespace graph_ops
