#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include "parallel_utils.hpp"
#include "wholememory/communicator.hpp"

#include "nvshmem.h"
#include "nvshmemx.h"
#include "wholememory/nvshmem_template.cuh"
#include "wholememory_test_utils.hpp"

#include <wholememory/device_reference.cuh>

__global__ void simple_shift(int* destination)
{
  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();
  int peer = (mype + 1) % npes;

  nvshmem_int_p(destination, mype, peer);
}

__global__ void simple_shift_p2p(int* destination)
{
  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();
  int peer = (mype + 1) % npes;

  // nvshmem_int_p(destination, mype, peer);
  int* dest = static_cast<int*>(nvshmem_ptr(destination, peer));
  dest[0]   = mype;
}

__global__ void read_next_rank_data(
  int* output, wholememory_gref_t global_tensor_ptr, int all_size_of_data, int rank, int world_size)
{
  size_t next_rank              = (rank + 1) % world_size;
  size_t eles_num_each_rank     = (all_size_of_data + world_size - 1) / world_size;
  size_t start_offest_next_rank = eles_num_each_rank * next_rank;
  size_t end_next_rank          = min<int>(eles_num_each_rank * (next_rank + 1), all_size_of_data);
  int eles_this_rank            = end_next_rank - start_offest_next_rank;
// printf("*****************in kernel  read_next_rank_data *************\n");
  const int stride = gridDim.x * blockDim.x;

  wholememory::device_reference<int> global_device_ref{global_tensor_ptr};

  for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < eles_this_rank; id += stride) {
    // output[id]=global_device_ref[start_offest_this_rank+id];
    output[id] = global_device_ref.load(start_offest_next_rank + id);
  }
}

void copy_host_array_to_wholememory_v2(void* host_array,
                                       wholememory_handle_t array_handle,
                                       wholememory_array_description_t array_desc,
                                       hipStream_t stream)
{
  void* local_array_ptr;
  size_t local_array_size, local_array_offset;
  EXPECT_EQ(wholememory_get_local_memory(
              &local_array_ptr, &local_array_size, &local_array_offset, array_handle),
            WHOLEMEMORY_SUCCESS);
  int64_t array_ele_size = wholememory_dtype_get_element_size(array_desc.dtype);
  EXPECT_EQ(local_array_size % array_ele_size, 0);
  EXPECT_EQ(local_array_offset % array_ele_size, 0);
  wholememory_comm_t wm_comm;
  EXPECT_EQ(wholememory_get_communicator(&wm_comm, array_handle), WHOLEMEMORY_SUCCESS);
  printf("****************local_array_ptr :%ld ,local_array_size : %ld*********************\n",
         local_array_ptr,
         local_array_size);
  if (local_array_size) {
    EXPECT_EQ(hipMemcpyAsync(local_array_ptr,
                              static_cast<char*>(host_array) + local_array_offset,
                              local_array_size,
                              hipMemcpyHostToDevice,
                              stream),
              hipSuccess);
    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
  }
  wholememory_communicator_barrier(wm_comm);
}

TEST(WholeMemoryNvshmemTest, SIMPLE_SHIFT)
{
  int dev_count = ForkGetDeviceCount();
  EXPECT_GE(dev_count, 1);
  WHOLEMEMORY_CHECK(dev_count >= 1);
  int nproc = dev_count;
  std::vector<std::array<int, 2>> pipes;
  CreatePipes(&pipes, dev_count);
  MultiProcessRun(nproc, [&pipes](int rank, int world_size) {
    // setenv("NVSHMEM_BOOTSTRAP","plugin",1);
    // setenv("NVSHMEM_BOOTSTRAP_PLUGIN","libnvshmem_wholememory_bootstrap.so",1);
    setenv("NVSHMEM_BOOTSTRAP", "mpi", 1);
    setenv("NVSHMEM_BOOTSTRAP_MPI_PLUGIN", "libnvshmem_wholememory_bootstrap.so", 1);
    EXPECT_EQ(hipSetDevice(rank), hipSuccess);

    wholememory_comm_t wm_comm1 = create_communicator_by_pipes(pipes, rank, world_size);
    EXPECT_EQ(wm_comm1->comm_id, 0);

    nvshmemx_init_attr_t attr;
    int mype, npes, mype_node;

    attr.mpi_comm = &wm_comm1;

    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    mype      = nvshmem_my_pe();
    npes      = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    EXPECT_EQ(mype, rank);
    EXPECT_EQ(npes, world_size);

    EXPECT_EQ(mype_node, rank);
    // EXPECT_EQ(hipSetDevice(rank), hipSuccess);
    hipStream_t stream;

    EXPECT_EQ(hipStreamCreate(&stream), hipSuccess);
    int* destination = (int*)nvshmem_malloc(sizeof(int));
    EXPECT_NE(destination, nullptr);
    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    int msg;
    EXPECT_EQ(hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream),
              hipSuccess);
    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    int expect_msg = (mype + npes - 1) % npes;
    EXPECT_EQ(msg, expect_msg);
    nvshmem_free(destination);
    nvshmem_finalize();
    EXPECT_EQ(hipStreamDestroy(stream), hipSuccess);

    EXPECT_EQ(wholememory::destroy_all_communicators(), WHOLEMEMORY_SUCCESS);

    WHOLEMEMORY_CHECK(::testing::Test::HasFailure() == false);
  });
}

TEST(WholeMemoryNvshmemTest, SIMPLE_SHIFT_P2P)
{
  int dev_count = ForkGetDeviceCount();
  EXPECT_GE(dev_count, 1);
  WHOLEMEMORY_CHECK(dev_count >= 1);
  int nproc = dev_count;
  std::vector<std::array<int, 2>> pipes;
  CreatePipes(&pipes, dev_count);
  MultiProcessRun(nproc, [&pipes](int rank, int world_size) {
    // setenv("NVSHMEM_BOOTSTRAP","plugin",1);
    // setenv("NVSHMEM_BOOTSTRAP_PLUGIN","libnvshmem_wholememory_bootstrap.so",1);
    setenv("NVSHMEM_BOOTSTRAP", "mpi", 1);
    setenv("NVSHMEM_BOOTSTRAP_MPI_PLUGIN", "libnvshmem_wholememory_bootstrap.so", 1);
    EXPECT_EQ(hipSetDevice(rank), hipSuccess);

    wholememory_comm_t wm_comm1 = create_communicator_by_pipes(pipes, rank, world_size);
    EXPECT_EQ(wm_comm1->comm_id, 0);

    nvshmemx_init_attr_t attr;
    int mype, npes, mype_node;

    attr.mpi_comm = &wm_comm1;

    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    mype      = nvshmem_my_pe();
    npes      = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    EXPECT_EQ(mype, rank);
    EXPECT_EQ(npes, world_size);

    EXPECT_EQ(mype_node, rank);
    // EXPECT_EQ(hipSetDevice(rank), hipSuccess);
    hipStream_t stream;

    EXPECT_EQ(hipStreamCreate(&stream), hipSuccess);
    int* destination = (int*)nvshmem_malloc(sizeof(int));
    EXPECT_NE(destination, nullptr);
    simple_shift_p2p<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    int msg;
    EXPECT_EQ(hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream),
              hipSuccess);
    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    int expect_msg = (mype + npes - 1) % npes;
    EXPECT_EQ(msg, expect_msg);
    nvshmem_free(destination);
    nvshmem_finalize();
    EXPECT_EQ(hipStreamDestroy(stream), hipSuccess);

    EXPECT_EQ(wholememory::destroy_all_communicators(), WHOLEMEMORY_SUCCESS);

    WHOLEMEMORY_CHECK(::testing::Test::HasFailure() == false);
  });
}

TEST(WholeMemoryNvshmemTest, NVSHMEM_TENSOR_GET)
{
  int dev_count = ForkGetDeviceCount();
  EXPECT_GE(dev_count, 1);
  WHOLEMEMORY_CHECK(dev_count >= 1);
  int nproc = dev_count;
  std::vector<std::array<int, 2>> pipes;
  CreatePipes(&pipes, dev_count);
  MultiProcessRun(nproc, [&pipes](int rank, int world_size) {


    // setenv("NVSHMEM_BOOTSTRAP","plugin",1);
    // setenv("NVSHMEM_BOOTSTRAP_PLUGIN","libnvshmem_wholememory_bootstrap.so",1);
    setenv("NVSHMEM_BOOTSTRAP", "mpi", 1);
    setenv("NVSHMEM_BOOTSTRAP_MPI_PLUGIN", "libnvshmem_wholememory_bootstrap.so", 1);
    EXPECT_EQ(hipSetDevice(rank), hipSuccess);

    wholememory_comm_t wm_comm1 = create_communicator_by_pipes(pipes, rank, world_size);
    EXPECT_EQ(wm_comm1->comm_id, 0);

    nvshmemx_init_attr_t attr;
    int mype, npes, mype_node;

    attr.mpi_comm = &wm_comm1;

    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    // TODO: run in multi node
    mype      = nvshmem_my_pe();
    npes      = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    EXPECT_EQ(mype, rank);
    EXPECT_EQ(npes, world_size);

    EXPECT_EQ(mype_node, rank);
    // EXPECT_EQ(hipSetDevice(rank), hipSuccess);
    hipStream_t stream;
    EXPECT_EQ(hipStreamCreate(&stream), hipSuccess);

    size_t tensor_size = 8192;
    EXPECT_EQ(tensor_size % world_size, 0);
    wholememory_dtype_t data_type = WHOLEMEMORY_DT_INT;
    auto memory_type              = WHOLEMEMORY_MT_NVSHMEM;
    auto memory_location          = WHOLEMEMORY_ML_DEVICE;

    auto array_desc = wholememory_create_array_desc(tensor_size, 0, data_type);

    wholememory_handle_t tensor_memory_handle;
    EXPECT_EQ(wholememory_malloc(&tensor_memory_handle,
                                 wholememory_get_memory_size_from_array(&array_desc),
                                 wm_comm1,
                                 memory_type,
                                 memory_location,
                                 wholememory_dtype_get_element_size(data_type)),
              WHOLEMEMORY_SUCCESS);
    EXPECT_NE(tensor_memory_handle, nullptr);
    std::vector<int> host_vec(tensor_size);
    size_t elements_each_rank = tensor_size / world_size;

    constexpr int OFFSET=100000;
    for (int i = 0; i < tensor_size; i++) {
      host_vec[i] = (i / elements_each_rank)*OFFSET+i;
    }

    printf("********************* tensor_memory_handle:%ld*************\n", tensor_memory_handle);
    copy_host_array_to_wholememory_v2(host_vec.data(), tensor_memory_handle, array_desc, stream);

    wholememory_gref_t wm_tensor_ptr_gref;

    EXPECT_EQ(wholememory_get_global_reference(&wm_tensor_ptr_gref, tensor_memory_handle),
              WHOLEMEMORY_SUCCESS);

    int len_this_rank = (tensor_size + world_size - 1) / world_size;
    int* output_d;
    EXPECT_EQ(hipMalloc(&output_d, sizeof(int) * len_this_rank), hipSuccess);
    read_next_rank_data<<<1, 256, 0, stream>>>(
      output_d, wm_tensor_ptr_gref, tensor_size, rank, world_size);

    int next_rank = (rank + 1) % world_size;

    std::vector<int> host_ref(len_this_rank);

    EXPECT_EQ(
      hipMemcpyAsync(
        host_ref.data(), output_d, sizeof(int) * len_this_rank, hipMemcpyDeviceToHost, stream),
      hipSuccess);
    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
    for (int i = 0; i < len_this_rank; i++) {
      EXPECT_EQ(host_ref[i], (next_rank*OFFSET+next_rank*len_this_rank+i));
    }

    EXPECT_EQ(wholememory_free(tensor_memory_handle), WHOLEMEMORY_SUCCESS);

    nvshmem_finalize();
    EXPECT_EQ(hipStreamDestroy(stream), hipSuccess);

    EXPECT_EQ(wholememory::destroy_all_communicators(), WHOLEMEMORY_SUCCESS);

    WHOLEMEMORY_CHECK(::testing::Test::HasFailure() == false);
  });
}
// ./gtests/WHOLEMEMORY_NVSHMEM_TEST --gtest_filter=*NVSHMEM_TENSOR_GET*



TEST(WholeMemoryNvshmemTest, NVSHMEM_TENSOR_GET_WITH_COMM)
{
  int dev_count = ForkGetDeviceCount();
  // int dev_count=1;
  EXPECT_GE(dev_count, 1);
  WHOLEMEMORY_CHECK(dev_count >= 1);
  int nproc = dev_count;
  std::vector<std::array<int, 2>> pipes;
  CreatePipes(&pipes, dev_count);
  MultiProcessRun(nproc, [&pipes](int rank, int world_size) {

    
    EXPECT_EQ(hipSetDevice(rank), hipSuccess);

    wholememory_comm_t wm_comm1 = create_communicator_by_pipes(pipes, rank, world_size);
    EXPECT_EQ(wm_comm1->comm_id, 0);

  
    // EXPECT_EQ(hipSetDevice(rank), hipSuccess);
    hipStream_t stream;
    EXPECT_EQ(hipStreamCreate(&stream), hipSuccess);

    size_t tensor_size = 8192;
    EXPECT_EQ(tensor_size % world_size, 0);
    wholememory_dtype_t data_type = WHOLEMEMORY_DT_INT;
    auto memory_type              = WHOLEMEMORY_MT_NVSHMEM;
    auto memory_location          = WHOLEMEMORY_ML_DEVICE;

    auto array_desc = wholememory_create_array_desc(tensor_size, 0, data_type);

    wholememory_handle_t tensor_memory_handle;
    EXPECT_EQ(wholememory_malloc(&tensor_memory_handle,
                                 wholememory_get_memory_size_from_array(&array_desc),
                                 wm_comm1,
                                 memory_type,
                                 memory_location,
                                 wholememory_dtype_get_element_size(data_type)),
              WHOLEMEMORY_SUCCESS);
    EXPECT_NE(tensor_memory_handle, nullptr);
    std::vector<int> host_vec(tensor_size);
    size_t elements_each_rank = tensor_size / world_size;

    constexpr int OFFSET=100000;
    for (int i = 0; i < tensor_size; i++) {
      host_vec[i] = (i / elements_each_rank)*OFFSET+i;
    }

    printf("********************* tensor_memory_handle:%ld*************\n", tensor_memory_handle);
    copy_host_array_to_wholememory_v2(host_vec.data(), tensor_memory_handle, array_desc, stream);

    wholememory_gref_t wm_tensor_ptr_gref;

    EXPECT_EQ(wholememory_get_global_reference(&wm_tensor_ptr_gref, tensor_memory_handle),
              WHOLEMEMORY_SUCCESS);

    int len_this_rank = (tensor_size + world_size - 1) / world_size;
    int* output_d;
    EXPECT_EQ(hipMalloc(&output_d, sizeof(int) * len_this_rank), hipSuccess);
    read_next_rank_data<<<1, 256, 0, stream>>>(
      output_d, wm_tensor_ptr_gref, tensor_size, rank, world_size);
    
    int next_rank = (rank + 1) % world_size;

    std::vector<int> host_ref(len_this_rank);
    WM_CUDA_CHECK(hipGetLastError());
    WM_CUDA_CHECK(
      hipMemcpyAsync(
        host_ref.data(), output_d, sizeof(int) * len_this_rank, hipMemcpyDeviceToHost, stream));
    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
    for (int i = 0; i < len_this_rank; i++) {
      EXPECT_EQ(host_ref[i], (next_rank*OFFSET+next_rank*len_this_rank+i));
    }

    EXPECT_EQ(wholememory_free(tensor_memory_handle), WHOLEMEMORY_SUCCESS);

    EXPECT_EQ(hipStreamDestroy(stream), hipSuccess);

    EXPECT_EQ(wholememory::destroy_all_communicators(), WHOLEMEMORY_SUCCESS);

    WHOLEMEMORY_CHECK(::testing::Test::HasFailure() == false);
  });
}

// ./gtests/WHOLEMEMORY_NVSHMEM_TEST --gtest_filter=*NVSHMEM_TENSOR_GET_WITH_COMM*
