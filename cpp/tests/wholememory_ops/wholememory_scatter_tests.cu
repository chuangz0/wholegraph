#include <gtest/gtest.h>

#include <wholememory/tensor_description.h>
#include <wholememory/wholememory.h>
#include <wholememory/wholememory_op.h>

#include "parallel_utils.hpp"
#include "wholememory/communicator.hpp"
#include "wholememory/env_func_ptrs.hpp"

#include "../wholememory/wholememory_test_utils.hpp"
#include "embedding_test_utils.hpp"

typedef struct WholeMemoryScatterTestParam {
  wholememory_matrix_description_t get_embedding_desc() const {
    int64_t matrix_sizes[2] = {embedding_dim, embedding_entry_count};
    return wholememory_create_matrix_desc(matrix_sizes, embedding_stride, embedding_storage_offset, embedding_type);
  }
  wholememory_array_description_t get_indices_desc() const {
    return wholememory_create_array_desc(indices_count, indices_storage_offset, indices_type);
  }
  wholememory_matrix_description_t get_input_desc() const {
    int64_t input_sizes[2] = {embedding_dim, indices_count};
    return wholememory_create_matrix_desc(input_sizes, input_stride, input_storage_offset, input_type);
  }
  int64_t get_embedding_granularity() const {
    return embedding_stride * wholememory_dtype_get_element_size(embedding_type);
  }

  WholeMemoryScatterTestParam& set_memory_type(wholememory_memory_type_t new_memory_type) {
    memory_type = new_memory_type;
    return *this;
  }
  WholeMemoryScatterTestParam& set_memory_location(wholememory_memory_location_t new_memory_location) {
    memory_location = new_memory_location;
    return *this;
  }
  WholeMemoryScatterTestParam& set_entry_count(int64_t entry_count) {
    embedding_entry_count = entry_count;
    return *this;
  }
  WholeMemoryScatterTestParam& set_embedding_dim(int64_t new_embedding_dim) {
    embedding_dim = new_embedding_dim;
    if (embedding_stride < embedding_dim) embedding_stride = embedding_dim;
    if (input_stride < embedding_dim) input_stride = embedding_dim;
    return *this;
  }
  WholeMemoryScatterTestParam& set_embedding_stride(int64_t new_embedding_stride) {
    embedding_stride = new_embedding_stride;
    return *this;
  }
  WholeMemoryScatterTestParam& set_input_stride(int64_t new_input_stride) {
    input_stride = new_input_stride;
    return *this;
  }
  WholeMemoryScatterTestParam& set_embedding_type(wholememory_dtype_t new_embedding_type) {
    embedding_type = new_embedding_type;
    return *this;
  }
  WholeMemoryScatterTestParam& set_indices_type(wholememory_dtype_t new_indices_type) {
    indices_type = new_indices_type;
    return *this;
  }
  WholeMemoryScatterTestParam& set_input_type(wholememory_dtype_t new_input_type) {
    input_type = new_input_type;
    return *this;
  }
  wholememory_memory_type_t memory_type = WHOLEMEMORY_MT_CHUNKED;
  wholememory_memory_location_t memory_location = WHOLEMEMORY_ML_DEVICE;
  int64_t embedding_entry_count = 1000000LL;
  int64_t embedding_dim = 32;
  int64_t embedding_stride = 32;
  int64_t indices_count = 100000;
  int64_t input_stride = 32;
  wholememory_dtype_t embedding_type = WHOLEMEMORY_DT_FLOAT;
  wholememory_dtype_t indices_type = WHOLEMEMORY_DT_INT;
  wholememory_dtype_t input_type = WHOLEMEMORY_DT_FLOAT;
  int64_t embedding_storage_offset = 0;
  int64_t indices_storage_offset = 0;
  int64_t input_storage_offset = 0;
} WholeMemoryScatterTestParam;

class WholeMemoryScatterParameterTests : public ::testing::TestWithParam<WholeMemoryScatterTestParam> {
};

TEST_P(WholeMemoryScatterParameterTests, ScatterTest) {
  auto params = GetParam();
  int dev_count = ForkGetDeviceCount();
  EXPECT_GE(dev_count, 1);
  std::vector<std::array<int, 2>> pipes;
  CreatePipes(&pipes, dev_count);
  MultiProcessRun(dev_count, [&params, &pipes](int world_rank, int world_size) {
    EXPECT_EQ(wholememory_init(0), WHOLEMEMORY_SUCCESS);

    EXPECT_EQ(hipSetDevice(world_rank), hipSuccess);

    wholememory_comm_t wm_comm = create_communicator_by_pipes(pipes, world_rank, world_size);

    wholememory_handle_t embedding_handle;
    auto embedding_desc = params.get_embedding_desc();
    auto indices_desc = params.get_indices_desc();
    auto input_desc = params.get_input_desc();
    size_t embedding_entry_size = params.get_embedding_granularity();
    EXPECT_EQ(wholememory_malloc(&embedding_handle,
                                 wholememory_get_memory_size_from_matrix(&embedding_desc),
                                 wm_comm,
                                 params.memory_type,
                                 params.memory_location,
                                 embedding_entry_size), WHOLEMEMORY_SUCCESS);

    hipStream_t stream;
    EXPECT_EQ(hipStreamCreate(&stream), hipSuccess);

    void  *host_indices, *dev_indices, *dev_input_buffer, *dev_gather_buffer, *host_gather_buffer, *host_input_buffer;
    size_t scatter_buffer_size = wholememory_get_memory_size_from_matrix(&input_desc);
    size_t indices_buffer_size = wholememory_get_memory_size_from_array(&indices_desc);

    EXPECT_EQ(hipHostMalloc(&host_indices, indices_buffer_size), hipSuccess);
    EXPECT_EQ(hipMalloc(&dev_indices, indices_buffer_size), hipSuccess);
    EXPECT_EQ(hipMalloc(&dev_input_buffer, scatter_buffer_size), hipSuccess);
    EXPECT_EQ(hipMalloc(&dev_gather_buffer, scatter_buffer_size), hipSuccess);
    EXPECT_EQ(hipHostMalloc(&host_input_buffer, scatter_buffer_size), hipSuccess);
    EXPECT_EQ(hipHostMalloc(&host_gather_buffer, scatter_buffer_size), hipSuccess);

    wholememory_ops::testing::host_random_init_indices(host_indices, indices_desc);
    EXPECT_EQ(hipMemcpyAsync(dev_indices,
                              host_indices,
                              wholememory_get_memory_size_from_array(&indices_desc),
                              hipMemcpyHostToDevice,
                              stream), hipSuccess);
    wholememory_ops::testing::device_get_expected_embedding(dev_input_buffer,
                                                            input_desc,
                                                            embedding_desc.dtype,
                                                            dev_indices,
                                                            indices_desc,
                                                            wholememory::get_default_env_func(),
                                                            stream);

    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
    wholememory_communicator_barrier(wm_comm);

    EXPECT_EQ(wholememory_scatter(dev_input_buffer, input_desc,
                                  dev_indices, indices_desc,
                                  embedding_handle, embedding_desc,
                                  wholememory::get_default_env_func(), stream), WHOLEMEMORY_SUCCESS);

    EXPECT_EQ(hipGetLastError(), hipSuccess);
    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
    wholememory_communicator_barrier(wm_comm);

    EXPECT_EQ(wholememory_gather(embedding_handle, embedding_desc,
                                 dev_indices, indices_desc,
                                 dev_gather_buffer, input_desc,
                                 wholememory::get_default_env_func(), stream), WHOLEMEMORY_SUCCESS);

    EXPECT_EQ(hipMemcpyAsync(host_gather_buffer,
                              dev_gather_buffer,
                              wholememory_get_memory_size_from_matrix(&input_desc),
                              hipMemcpyDeviceToHost,
                              stream), hipSuccess);
    EXPECT_EQ(hipMemcpyAsync(host_input_buffer,
                              dev_input_buffer,
                              wholememory_get_memory_size_from_matrix(&input_desc),
                              hipMemcpyDeviceToHost,
                              stream), hipSuccess);
    EXPECT_EQ(hipGetLastError(), hipSuccess);
    EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);

    wholememory_ops::testing::host_check_embedding_same(host_gather_buffer,
                                                        input_desc,
                                                        host_input_buffer,
                                                        input_desc);

    EXPECT_EQ(hipHostFree(host_indices), hipSuccess);
    EXPECT_EQ(hipFree(dev_indices), hipSuccess);
    EXPECT_EQ(hipFree(dev_input_buffer), hipSuccess);
    EXPECT_EQ(hipFree(dev_gather_buffer), hipSuccess);
    EXPECT_EQ(hipHostFree(host_input_buffer), hipSuccess);
    EXPECT_EQ(hipHostFree(host_gather_buffer), hipSuccess);

    EXPECT_EQ(wholememory_free(embedding_handle), WHOLEMEMORY_SUCCESS);

    EXPECT_EQ(wholememory::destroy_all_communicators(), WHOLEMEMORY_SUCCESS);

    EXPECT_EQ(wholememory_finalize(), WHOLEMEMORY_SUCCESS);
  });
}

INSTANTIATE_TEST_CASE_P(
    WholeMemoryScatterOpTests,
    WholeMemoryScatterParameterTests,
    ::testing::Values(
#if 1
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_memory_location(WHOLEMEMORY_ML_HOST),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_memory_location(WHOLEMEMORY_ML_HOST),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_memory_location(WHOLEMEMORY_ML_HOST),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_dim(128),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_dim(128),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_dim(128),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_dim(127),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_dim(127),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_dim(127),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_dim(129),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_dim(129),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_dim(129),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_dim(513),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_dim(513),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_dim(513),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_input_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_input_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_input_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_type(WHOLEMEMORY_DT_HALF).set_input_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_type(WHOLEMEMORY_DT_HALF).set_input_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_type(WHOLEMEMORY_DT_HALF).set_input_type(WHOLEMEMORY_DT_HALF),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_indices_type(WHOLEMEMORY_DT_INT64),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_indices_type(WHOLEMEMORY_DT_INT64),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_indices_type(WHOLEMEMORY_DT_INT64),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_input_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_input_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_input_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CONTINUOUS).set_embedding_type(WHOLEMEMORY_DT_HALF).set_embedding_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_CHUNKED).set_embedding_type(WHOLEMEMORY_DT_HALF).set_embedding_stride(33),
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED).set_embedding_type(WHOLEMEMORY_DT_HALF).set_embedding_stride(33),
#endif
        WholeMemoryScatterTestParam().set_memory_type(WHOLEMEMORY_MT_DISTRIBUTED)
    ));

