#include <error.hpp>
#include <gtest/gtest.h>

#include "./spmm_csr_no_weight_utils.hpp"
#include "gspmm_csr_weighted_test_utils.hpp"
#include "wholememory/initialize.hpp"
#include <wholememory/graph_op.h>
#include <wholememory/tensor_description.h>

typedef struct gSpMMCsrWeightedForwardTestParam {
  gSpMMCsrWeightedForwardTestParam& set_graph_row_num(int new_graph_row_num)
  {
    graph_row_num = new_graph_row_num;
    return *this;
  }
  gSpMMCsrWeightedForwardTestParam& set_graph_col_num(int new_graph_col_num)
  {
    graph_col_num = new_graph_col_num;
    return *this;
  }
  gSpMMCsrWeightedForwardTestParam& set_graph_edge_num(int new_graph_edge_num)
  {
    graph_edge_num = new_graph_edge_num;
    return *this;
  }
  gSpMMCsrWeightedForwardTestParam& set_num_heads(int new_num_heads)
  {
    num_heads = new_num_heads;
    return *this;
  }
  gSpMMCsrWeightedForwardTestParam& set_feature_dim(int new_feature_dim)
  {
    feature_dim = new_feature_dim;
    return *this;
  }
  gSpMMCsrWeightedForwardTestParam& set_weight_dtype(wholememory_dtype_t new_weight_dtype)
  {
    weight_dtype = new_weight_dtype;
    return *this;
  }

  wholememory_array_description_t get_csr_row_ptr_array_desc()
  {
    return wholememory_create_array_desc(graph_row_num + 1, 0, WHOLEMEMORY_DT_INT);
  }
  wholememory_array_description_t get_csr_col_ptr_array_desc()
  {
    return wholememory_create_array_desc(graph_edge_num, 0, WHOLEMEMORY_DT_INT);
  }

  int get_graph_row_num() const { return graph_row_num; }
  int get_graph_col_num() const { return graph_col_num; }
  int get_graph_edge_num() const { return graph_edge_num; }
  int get_num_heads() const { return num_heads; }
  int get_feature_dim() const { return feature_dim; }
  wholememory_dtype_t get_weight_dtype() const { return weight_dtype; }

  int graph_row_num                = 3;
  int graph_col_num                = 4;
  int graph_edge_num               = 10;
  int num_heads                    = 8;
  int feature_dim                  = 8;
  wholememory_dtype_t weight_dtype = WHOLEMEMORY_DT_FLOAT;
} gSpMMCsrWeightedForwardTestParam;

class gSpMMCsrWeightedForwardParameterTests
  : public ::testing::TestWithParam<gSpMMCsrWeightedForwardTestParam> {};

TEST_P(gSpMMCsrWeightedForwardParameterTests, gSpmmCsrWeightedForwardParameterTest)
{
  auto params = GetParam();
  int dev_count;
  EXPECT_EQ(hipGetDeviceCount(&dev_count), hipSuccess);
  EXPECT_GE(dev_count, 1);

  hipStream_t stream;
  EXPECT_EQ(hipStreamCreate(&stream), hipSuccess);

  auto graph_row_num          = params.get_graph_row_num();
  auto graph_col_num          = params.get_graph_col_num();
  auto graph_edge_num         = params.get_graph_edge_num();
  auto num_heads              = params.get_num_heads();
  auto feature_dim            = params.get_feature_dim();
  auto weight_type            = params.get_weight_dtype();
  auto csr_row_ptr_array_desc = params.get_csr_row_ptr_array_desc();
  auto csr_col_ptr_array_desc = params.get_csr_col_ptr_array_desc();
  void* host_csr_row_ptr =
    (void*)malloc(wholememory_get_memory_size_from_array(&csr_row_ptr_array_desc));
  void* host_csr_col_ptr =
    (void*)malloc(wholememory_get_memory_size_from_array(&csr_col_ptr_array_desc));
  graph_ops::testing::gen_local_csr_graph(graph_row_num,
                                          graph_col_num,
                                          graph_edge_num,
                                          host_csr_row_ptr,
                                          csr_row_ptr_array_desc,
                                          host_csr_col_ptr,
                                          csr_col_ptr_array_desc);
  wholememory_tensor_description_t csr_row_ptr_tensor_desc, csr_col_ptr_tensor_desc,
    edge_weight_tensor_desc, feature_tensor_desc, output_feature_tensor_desc;
  wholememory_copy_array_desc_to_tensor(&csr_row_ptr_tensor_desc, &csr_row_ptr_array_desc);
  wholememory_copy_array_desc_to_tensor(&csr_col_ptr_tensor_desc, &csr_col_ptr_array_desc);
  wholememory_initialize_tensor_desc(&edge_weight_tensor_desc);
  edge_weight_tensor_desc.dim            = 2;
  edge_weight_tensor_desc.sizes[0]       = graph_edge_num;
  edge_weight_tensor_desc.sizes[1]       = num_heads;
  edge_weight_tensor_desc.strides[0]     = num_heads;
  edge_weight_tensor_desc.strides[1]     = 1;
  edge_weight_tensor_desc.storage_offset = 0;
  edge_weight_tensor_desc.dtype          = weight_type;
  wholememory_initialize_tensor_desc(&feature_tensor_desc);
  feature_tensor_desc.dim            = 3;
  feature_tensor_desc.sizes[0]       = graph_col_num;
  feature_tensor_desc.sizes[1]       = num_heads;
  feature_tensor_desc.sizes[2]       = feature_dim;
  feature_tensor_desc.strides[0]     = feature_dim * num_heads;
  feature_tensor_desc.strides[1]     = feature_dim;
  feature_tensor_desc.strides[2]     = 1;
  feature_tensor_desc.storage_offset = 0;
  feature_tensor_desc.dtype          = weight_type;

  wholememory_initialize_tensor_desc(&output_feature_tensor_desc);
  output_feature_tensor_desc.dim            = 3;
  output_feature_tensor_desc.sizes[0]       = graph_row_num;
  output_feature_tensor_desc.sizes[1]       = num_heads;
  output_feature_tensor_desc.sizes[2]       = feature_dim;
  output_feature_tensor_desc.strides[0]     = feature_dim * num_heads;
  output_feature_tensor_desc.strides[1]     = feature_dim;
  output_feature_tensor_desc.strides[2]     = 1;
  output_feature_tensor_desc.storage_offset = 0;
  output_feature_tensor_desc.dtype          = weight_type;

  void* host_edge_weight_ptr =
    (void*)malloc(wholememory_get_memory_size_from_tensor(&edge_weight_tensor_desc));
  void* host_feature_ptr =
    (void*)malloc(wholememory_get_memory_size_from_tensor(&feature_tensor_desc));
  void* host_output_feature_ptr =
    (void*)malloc(wholememory_get_memory_size_from_tensor(&output_feature_tensor_desc));
  void* host_ref_output_feature_ptr =
    (void*)malloc(wholememory_get_memory_size_from_tensor(&output_feature_tensor_desc));
  graph_ops::testing::gen_features(host_edge_weight_ptr, edge_weight_tensor_desc);
  graph_ops::testing::gen_features(host_feature_ptr, feature_tensor_desc);
  void *dev_csr_row_ptr, *dev_csr_col_ptr, *dev_edge_weight_ptr, *dev_feature_ptr,
    *dev_output_feature_ptr;
  EXPECT_EQ(
    hipMalloc(&dev_csr_row_ptr, wholememory_get_memory_size_from_array(&csr_row_ptr_array_desc)),
    hipSuccess);
  EXPECT_EQ(
    hipMalloc(&dev_csr_col_ptr, wholememory_get_memory_size_from_array(&csr_col_ptr_array_desc)),
    hipSuccess);
  EXPECT_EQ(hipMalloc(&dev_edge_weight_ptr,
                       wholememory_get_memory_size_from_tensor(&edge_weight_tensor_desc)),
            hipSuccess);
  EXPECT_EQ(
    hipMalloc(&dev_feature_ptr, wholememory_get_memory_size_from_tensor(&feature_tensor_desc)),
    hipSuccess);
  EXPECT_EQ(hipMalloc(&dev_output_feature_ptr,
                       wholememory_get_memory_size_from_tensor(&output_feature_tensor_desc)),
            hipSuccess);

  EXPECT_EQ(hipMemcpy(dev_csr_row_ptr,
                       host_csr_row_ptr,
                       wholememory_get_memory_size_from_array(&csr_row_ptr_array_desc),
                       hipMemcpyHostToDevice),
            hipSuccess);

  EXPECT_EQ(hipMemcpy(dev_csr_col_ptr,
                       host_csr_col_ptr,
                       wholememory_get_memory_size_from_array(&csr_col_ptr_array_desc),
                       hipMemcpyHostToDevice),
            hipSuccess);

  EXPECT_EQ(hipMemcpy(dev_edge_weight_ptr,
                       host_edge_weight_ptr,
                       wholememory_get_memory_size_from_tensor(&edge_weight_tensor_desc),
                       hipMemcpyHostToDevice),
            hipSuccess);

  EXPECT_EQ(hipMemcpy(dev_feature_ptr,
                       host_feature_ptr,
                       wholememory_get_memory_size_from_tensor(&feature_tensor_desc),
                       hipMemcpyHostToDevice),
            hipSuccess);

  wholememory_tensor_t csr_row_ptr_tensor, csr_col_ptr_tensor, edge_weight_tensor, feature_tensor,
    output_feature_tensor;
  EXPECT_EQ(wholememory_make_tensor_from_pointer(
              &csr_row_ptr_tensor, dev_csr_row_ptr, &csr_row_ptr_tensor_desc),
            WHOLEMEMORY_SUCCESS);
  EXPECT_EQ(wholememory_make_tensor_from_pointer(
              &csr_col_ptr_tensor, dev_csr_col_ptr, &csr_col_ptr_tensor_desc),
            WHOLEMEMORY_SUCCESS);

  EXPECT_EQ(wholememory_make_tensor_from_pointer(
              &edge_weight_tensor, dev_edge_weight_ptr, &edge_weight_tensor_desc),
            WHOLEMEMORY_SUCCESS);

  EXPECT_EQ(
    wholememory_make_tensor_from_pointer(&feature_tensor, dev_feature_ptr, &feature_tensor_desc),
    WHOLEMEMORY_SUCCESS);
  EXPECT_EQ(wholememory_make_tensor_from_pointer(
              &output_feature_tensor, dev_output_feature_ptr, &output_feature_tensor_desc),
            WHOLEMEMORY_SUCCESS);
  EXPECT_EQ(gspmm_csr_weighted_forward(csr_row_ptr_tensor,
                                       csr_col_ptr_tensor,
                                       edge_weight_tensor,
                                       feature_tensor,
                                       output_feature_tensor,
                                       stream),
            WHOLEMEMORY_SUCCESS);
  EXPECT_EQ(hipMemcpyAsync(host_output_feature_ptr,
                            dev_output_feature_ptr,
                            wholememory_get_memory_size_from_tensor(&output_feature_tensor_desc),
                            hipMemcpyDeviceToHost,
                            stream),
            hipSuccess);
  EXPECT_EQ(hipStreamSynchronize(stream), hipSuccess);
  graph_ops::testing::host_gspmm_csr_weighted_forward(host_csr_row_ptr,
                                                      csr_row_ptr_array_desc,
                                                      host_csr_col_ptr,
                                                      csr_col_ptr_array_desc,
                                                      host_edge_weight_ptr,
                                                      edge_weight_tensor_desc,
                                                      host_feature_ptr,
                                                      feature_tensor_desc,
                                                      host_ref_output_feature_ptr,
                                                      output_feature_tensor_desc);
  graph_ops::testing::host_check_float_tensor_same(host_output_feature_ptr,
                                                   output_feature_tensor_desc,
                                                   host_ref_output_feature_ptr,
                                                   output_feature_tensor_desc);

  EXPECT_EQ(hipFree(dev_csr_row_ptr), hipSuccess);
  EXPECT_EQ(hipFree(dev_csr_col_ptr), hipSuccess);
  EXPECT_EQ(hipFree(dev_edge_weight_ptr), hipSuccess);
  EXPECT_EQ(hipFree(dev_feature_ptr), hipSuccess);
  EXPECT_EQ(hipFree(dev_output_feature_ptr), hipSuccess);
  if (host_csr_row_ptr != nullptr) free(host_csr_row_ptr);
  if (host_csr_col_ptr != nullptr) free(host_csr_col_ptr);
  if (host_edge_weight_ptr != nullptr) free(host_edge_weight_ptr);
  if (host_feature_ptr != nullptr) free(host_feature_ptr);
  if (host_output_feature_ptr != nullptr) free(host_output_feature_ptr);
  if (host_ref_output_feature_ptr != nullptr) free(host_ref_output_feature_ptr);

  EXPECT_EQ(hipStreamDestroy(stream), hipSuccess);

  WHOLEMEMORY_CHECK(::testing::Test::HasFailure() == false);
}

INSTANTIATE_TEST_SUITE_P(gSpmmCsrWeightFrowardTests,
                         gSpMMCsrWeightedForwardParameterTests,
                         ::testing::Values(gSpMMCsrWeightedForwardTestParam()
                                             .set_graph_row_num(1025)
                                             .set_graph_col_num(2379)
                                             .set_graph_edge_num(10793)
                                             .set_num_heads(32)
                                             .set_feature_dim(128)
                                             .set_weight_dtype(WHOLEMEMORY_DT_FLOAT)));